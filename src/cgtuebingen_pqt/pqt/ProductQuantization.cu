#include "hip/hip_runtime.h"
#ifndef NEARESTNEIGHBOR_PRODUCTQUANTIZATION_C
#define NEARESTNEIGHBOR_PRODUCTQUANTIZATION_C

#include "ProductQuantization.hh"

#define OUTPUT
#include "helper.hh"

using namespace std;


using namespace pqt;

namespace pqtPQ {

/** default constructor */

ProductQuantization::ProductQuantization(uint _dim, uint _p) :
		d_dim(_dim), d_codeBook(NULL), d_p(_p) {
}

ProductQuantization::~ProductQuantization() {
	if (d_codeBook)
		hipFree(d_codeBook);
}

__device__ float sqr(const float &x) {
	return x * x;
}

/** for all vectors in A compute distance to all vectors in B of dimension _dim
 * computes the partial sum blockDim.x columns
 * The kernel produces gridDim.z distance measures per vector per.
 */

__global__ void calcDistKernel(float* _res, const float * _A, const float * _B,
		uint _Arows, uint _Brows, uint _dim) {

	extern __shared__ float shm[];

	float *Ablock = shm;
	float *Bblock = shm + blockDim.x * blockDim.y;
	float *AB = Bblock + blockDim.x * blockDim.y;

	uint id = threadIdx.x + threadIdx.y * blockDim.x;
	uint arow = threadIdx.y + blockIdx.y * blockDim.y;
	uint brow = threadIdx.y + blockIdx.x * blockDim.x;
	uint ocol = threadIdx.x + blockIdx.x * blockDim.x;

	uint AOffs = threadIdx.y * blockDim.x;
	uint BOffs = threadIdx.x * blockDim.x;

	AB[id] = 0.;

	int j = blockIdx.z;
	{
//	for (int j = 0; j < _Acols; j += blockDim.x) {
		// load block of A and B
		uint col = threadIdx.x + j * blockDim.x;

		Bblock[id] = 0.;
		Ablock[id] = 0.;
		if (col < _dim) {
			if (brow < _Brows)
				Bblock[id] = _B[brow * _dim + col];
			if (arow < _Arows)
				Ablock[id] = _A[arow * _dim + col];
		}
		__syncthreads();

//		if ((col < _Acols) && (arow < _Arows))
//			printf(" A B %i %f %f \n", id, Ablock[id], Bblock[id]);

		// compute partial differences
		for (int i = 0; i < blockDim.x; i++) {
			AB[id] += sqr(Ablock[AOffs + i] - Bblock[BOffs + i]);
		}
		__syncthreads();

	}

// write out the result
	if ((arow < _Arows) && (ocol < _Brows)) {
		//	_res[arow][ocol] += AB[id];
		*(_res + j * _Arows * _Brows + (arow * _Brows + ocol)) = AB[id];
//			printf(" AB %i %i %i %f \n", id, arow, ocol, AB[id]);
	}

}

// currently fixed block size of 16
void ProductQuantization::calcDist(float* _res, const float* _A,
		const float* _B, uint _Arows, uint _Brows, uint _dim) const {

	const uint blockSize = 16;

	dim3 block(blockSize, blockSize, 1);

	uint shmSize = (blockSize * blockSize * 3) * sizeof(float);
//cout << "requested shm: " << shmSize << endl;

	hipMemset(_res, 0, _Arows * _Brows * sizeof(float));

	dim3 grid(idiv(_Brows, blockSize), idiv(_Arows, blockSize),
			idiv(_dim, blockSize));

	calcDistKernel<<<grid, block, shmSize>>>(_res, _A, _B, _Arows, _Brows,
			_dim);
	checkCudaErrors(hipDeviceSynchronize());

//	outputMat("dist: ", _res, 10, 10);

}

/** blockd Id.x corresponds to the cluster center, blockId.y is used to span multiple kernels
 * will update the codebook vector of this center. As each y block is only adding some of the input vectors the last block is normalizing the vector
 * Assumes that blockDim.x corresponds to the length of the product snippets
 */__global__ void avgClusterKernel(float* _codebook, float * _count,
		uint *_retirementCount, uint _yElem, uint _dim, const float * _A,
		uint _N, uint _p, uint* _assignd) {

	__shared__ bool amLast;
	extern __shared__ float shm[];

	float count = 0;

	for (int i = threadIdx.x; i < _dim; i += blockDim.x) {
		shm[i] = 0.;
	}

	int stop = (blockIdx.y + 1) * _yElem;
	stop = (stop < _N) ? stop : _N;
	// accumulate the vectors that belong to this cluster center
	for (int n = blockIdx.y * _yElem; n < stop; n++) {
		uint p = 0;
		const float *v = _A + n * _dim;
		for (int i = threadIdx.x; i < _dim; i += blockDim.x, p++) {
			if (_assignd[p * _N + n] == blockIdx.x) {
				shm[i] += v[i];

				if (threadIdx.x == p)
					count++;
			}
		}
	}

	// store the result
	__syncthreads();
	for (int i = threadIdx.x; i < _dim; i += blockDim.x) {
		atomicAdd(_codebook + blockIdx.x * _dim + i, shm[i]);
	}
	__threadfence();

	// write out the p counts
	if (threadIdx.x < _p)
		atomicAdd(_count + threadIdx.x * gridDim.x + blockIdx.x, count);

	if (threadIdx.x == 0) {
		uint ticket = atomicInc(_retirementCount + blockIdx.x, gridDim.y);
		// If the ticket ID is equal to the number of blocks, we are the last block!
		amLast = (ticket == gridDim.y - 1);
	}
	__syncthreads();

	// the last block is responsible for dividing by the number of vectors added to this center
	if (amLast) {
		uint p = 0;
		for (int i = threadIdx.x; i < _dim; i += blockDim.x, p++) {
			float count = _count[p * gridDim.x + blockIdx.x];
			count = (count == 0.) ? 1. : count;
			_codebook[blockIdx.x * _dim + i] /= count;
		}
		// reset retirement count for next iteration
		if (threadIdx.x == 0) {
			_retirementCount[blockIdx.x] = 0;
		}
	}
}

__global__ void assignKernel(uint* _assign, uint _N, const float* _distMat,
		uint _nClusters, uint _p) {
	extern __shared__ float shm[];

	float* sharedMin = shm;
	uint* sharedIdx = (uint*) (shm + blockDim.x);
	// initialize with first element

	for (int row = blockIdx.x; row < _N; row += gridDim.x) {
		for (int p = 0; p < _p; p++) {
			const float* matRow = _distMat + p * _nClusters * _N
					+ row * _nClusters;
			float vMin = matRow[0];
			uint minIdx = 0;

			for (uint i = threadIdx.x; i < _nClusters; i += blockDim.x) {
				float vMin2 = matRow[i];
				if (vMin2 < vMin) {
					vMin = vMin2;
					minIdx = i;
				}
			}

			sharedMin[threadIdx.x] = vMin;
			sharedIdx[threadIdx.x] = minIdx;

			for (uint stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
				__syncthreads();
				if (threadIdx.x < stride) {
					float vMin2 = sharedMin[threadIdx.x + stride];
					if (vMin2 < vMin) {
						vMin = vMin2;
						sharedMin[threadIdx.x] = vMin;
						sharedIdx[threadIdx.x] =
								sharedIdx[threadIdx.x + stride];
					}
				}
			}
			__syncthreads();

			if (threadIdx.x == 0) {
				_assign[p * _N + row] = sharedIdx[0];
			}
		}
	}
}

void ProductQuantization::getAssignment(uint* _assignd, const float* _distMat,
		uint _N, uint _nClusters) const {
	// perform a minimum reduction per vector _N
	dim3 block;
	setReductionBlocks(block, _nClusters);

	uint nBlocks = (_N < MAX_BLOCKS) ? _N : MAX_BLOCKS;
	dim3 grid(nBlocks, 1, 1);

	cout << "block: " << block.x << " " << block.y << "  " << block.z << endl;

	uint shmSize = block.x * 2 * sizeof(float);
	assignKernel<<<grid, block, shmSize>>>(_assignd, _N, _distMat, _nClusters,
			d_p);
	checkCudaErrors(hipDeviceSynchronize());
}

__global__ void splitVectorKernel(float* _codeBook, uint _dim, uint _nClusters,
		float _epsilon) {

	uint idx = blockIdx.x * _dim + threadIdx.x;
	float orig = _codeBook[idx];

	_codeBook[idx] = orig * (1. + _epsilon);

	idx += _nClusters * _dim;

	_codeBook[idx] = orig * (1. - _epsilon);

}

void ProductQuantization::splitCodeBook(uint &_nClusters, float _epsilon) {

	dim3 block(d_dim, 1, 1);

	dim3 grid(_nClusters, 1, 1);

	splitVectorKernel<<<grid, block>>>(d_codeBook, d_dim, _nClusters, _epsilon);

	_nClusters *= 2;

}

void ProductQuantization::createCodeBook(uint _k, const float* _A, uint _N) {

	uint *assign = new uint[_N];
	uint *old_assign = new uint[_N];

	uint *assignd;
	float* countd;
	uint* retirementCountd;
	float* distd;
	float* maxRadd;

	hipMalloc(&assignd, _N * sizeof(uint));

	hipMalloc(&d_codeBook, _k * d_dim * sizeof(float));
	hipMalloc(&countd, _k * d_p * sizeof(float));
	hipMalloc(&retirementCountd, _k * sizeof(uint));
	hipMalloc(&distd, _k * d_p * _N * sizeof(float));
	hipMalloc(&maxRadd, _k * d_p * sizeof(float));

	uint nClusters = 1;
	// initialize to get the first cluster average
	hipMemset(assignd, 0, _N * sizeof(uint));

	hipMemset(retirementCountd, 0, _k * sizeof(uint));
	hipMemset(countd, 0, _k * d_p * sizeof(int));
	hipMemset(d_codeBook, 0, d_dim * sizeof(float));

	uint yElem = 16;

	//dim3 block(d_dim, 1, 1);
	dim3 block(16, 1, 1);
	dim3 grid(nClusters, idiv(_N, yElem), 1);

	size_t shmSize = d_dim * sizeof(float);
	avgClusterKernel<<<grid, block, shmSize>>>(d_codeBook, countd,
			retirementCountd, yElem, d_dim, _A, _N, d_p, assignd);

	float epsilon = 0.0001;

	while (nClusters < _k) {

		splitCodeBook(nClusters, epsilon);
		cout << "nClusters" << nClusters << endl;

		uint converged = 0;

		do {

			hipMemset(countd, 0, _k * d_p * sizeof(int));

			calcDist(distd, _A, d_codeBook, _N, nClusters, d_dim);

			getAssignment(assignd, distd, _N, nClusters);

//			getMaxRad(maxRadd, nClusters, assignd, _N, distd);
			//outputVecUint("Assign", assignd, _N);

			uint yElem = 256;

			dim3 block(d_dim, 1, 1);
			dim3 grid(nClusters, idiv(_N, yElem), 1);

			size_t shmSize = d_dim * sizeof(float);
			hipMemset(d_codeBook, 0, nClusters * d_dim * sizeof(float));
			avgClusterKernel<<<grid, block, shmSize>>>(d_codeBook, countd,
					retirementCountd, yElem, d_dim, _A, _N, d_p, assignd);

			cout << nClusters << endl;
			outputVec("count:", countd, nClusters);
			//outputVec("avg: ", d_codeBook, d_dim);

			hipMemcpy(assign, assignd, _N * sizeof(uint),
					hipMemcpyDeviceToHost);
			converged = 0;
			for (int i = 0; i < _N; i++) {
				if (assign[i] != old_assign[i]) {
					converged++;
				}
			}
			memcpy(old_assign, assign, _N * sizeof(uint));

			cout << "non- converged" << converged << endl;
		} while (converged > 0.001 * _N);

		//outputMat("dist:", distd, _N, nClusters);
		outputMat("codebook", d_codeBook, nClusters, d_dim );

//		getMaxRad(maxRadd, nClusters, assignd, _N, distd);

//		calcDist(distd, d_codeBook, d_codeBook, nClusters, nClusters, d_dim);

//		outputMat("codebook Dist", distd, nClusters, nClusters);

//		char c;
//		cin >> c;

	}

	hipFree(distd);
	hipFree(countd);
	hipFree(retirementCountd);
	hipFree(assignd);

	delete[] old_assign;
	delete[] assign;
}

} /* namespace */

#endif /* NEARESTNEIGHBOR_PRODUCTQUANTIZATION_C */
