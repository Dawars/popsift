/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */

#include "assist.h"
#include "debug_macros.h"
#include "plane_2d.h"

#include <hip/hip_runtime.h>

#include <cstdlib>
#include <cstring>
#include <iostream>
#ifndef _WIN32
#include <unistd.h>
#else
#define NOMINMAX
#define WIN32_LEAN_AND_MEAN
#include <Windows.h>
#include <malloc.h>
#endif



using namespace std;

namespace popsift {

__host__
void* PlaneBase::allocDev2D( size_t& pitch, int w, int h, int elemSize, PlaneMapMode m )
{
    _mode = m;

    // cerr << "Alloc " << w*h*elemSize << " B" << endl;
    void*       ptr;
    hipError_t err;

    std::cerr << "Allocating 2D plane, elem size " << elemSize
              << " size " << w << "x" << h
              << " type " << (m==OnDevice?"OnDevice":"ManagedMen")
              << " pitch " << pitch
              << std::endl;

    if( m == OnDevice )
    {
        std::cerr << "    Trying to allocate " << w * elemSize << "x" << h << " bytes" << std::endl;
        err = hipMallocPitch( &ptr, &pitch, w * elemSize, h );
        POP_CUDA_FATAL_TEST( err, "Cannot allocate pitched CUDA memory: " );
        std::cerr << "    Allocated " << pitch << "x" << h << "=" << pitch*h << " bytes" << std::endl;
        return ptr;
    }
    else if( m == ManagedMem )
    {
        size_t sz = w * elemSize;
        std::cerr << "    Trying to allocate " << w * elemSize << "x" << h << " bytes with alignment " << pitch << std::endl;
        size_t rest = sz % pitch;
        if( rest == 0 )
            pitch = sz;
        else
            pitch = sz + pitch - rest;
        std::cerr << "    Trying to allocate " << pitch << "x" << h << "=" << pitch*h << " bytes" << std::endl;
        sz  = pitch * h;
        err = hipMallocManaged( &ptr, sz );
        POP_CUDA_FATAL_TEST( err, "Cannot allocate managed CUDA memory: " );
        std::cerr << "    Allocated " << pitch << "x" << h << "=" << pitch*h << " bytes" << std::endl;
        return ptr;
    }
    else
    {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Allocation mode not correct in device plane allocation" << endl;
        exit( -1 );
    }
}

__host__
void* PlaneBase::allocHost2D( int w, int h, int elemSize, PlaneMapMode m )
{
    _mode = m;

    int sz = w * h * elemSize;

    if( m == Unaligned )
    {
        void* ptr = malloc( sz );

        if( ptr != 0 ) return ptr;
        
#ifdef _GNU_SOURCE
        char b[100];
        const char* buf = strerror_r( errno, b, 100 );
#else
        const char *buf = strerror(errno);
#endif
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to allocate " << sz << " bytes of unaligned host memory." << endl
             << "    Cause: " << buf << endl;
        exit( -1 );
    }
    else if( m == PageAligned )
    {
        void* ptr = memalign(getPageSize(), sz);
        if(ptr)
            return ptr;

#ifdef _GNU_SOURCE
        char b[100];
        const char* buf = strerror_r( errno, b, 100 );
#else
		const char* buf = strerror(errno);
#endif
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Failed to allocate " << sz << " bytes of page-aligned host memory." << endl
             << "    Cause: " << buf << endl
             << "    Trying to allocate unaligned instead." << endl;

        return allocHost2D( w, h, elemSize, Unaligned );
    }
    else if( m == CudaAllocated )
    {
        void* ptr;
        hipError_t err;
        err = hipHostMalloc( &ptr, sz );
        POP_CUDA_FATAL_TEST( err, "Failed to allocate aligned and pinned host memory: " );
        return ptr;
    }
    else
    {
        cerr << __FILE__ << ":" << __LINE__ << endl
             << "    Allocation mode not correct in host plane allocation" << endl;
        exit( -1 );
    }
}

__host__
void PlaneBase::free2D( void* data )
{
    if (!data)
        return;

    switch( _mode )
    {
    case OnDevice :
    case ManagedMem :
        hipFree(data);
        return;
    case CudaAllocated :
        hipHostFree(data);
        return;
    case Unaligned :
        free(data);
        return;
    case PageAligned :
        memalign_free( data );
        return;
    default :
        assert(!"Invalid PlaneMapMode");
    }
}

__host__
void PlaneBase::memcpyToDevice( void* dst, int dst_pitch,
                                void* src, int src_pitch,
                                short cols, short rows,
                                int elemSize )
{
    assert( dst != 0 );
    assert( dst_pitch != 0 );
    assert( src != 0 );
    assert( src_pitch != 0 );
    assert( cols != 0 );
    assert( rows != 0 );
    hipError_t err;
    err = hipMemcpy2D( dst, dst_pitch,
                        src, src_pitch,
                        cols*elemSize, rows,
                        hipMemcpyHostToDevice );
    POP_CUDA_FATAL_TEST( err, "Failed to copy 2D plane host-to-device: " );
}

__host__
void PlaneBase::memcpyToDevice( void* dst, int dst_pitch,
                                void* src, int src_pitch,
                                short cols, short rows,
                                int elemSize,
                                hipStream_t stream )
{
    assert( dst != 0 );
    assert( dst_pitch != 0 );
    assert( src != 0 );
    assert( src_pitch != 0 );
    assert( cols != 0 );
    assert( rows != 0 );
    hipError_t err;
    err = hipMemcpy2DAsync( dst, dst_pitch,
                             src, src_pitch,
                             cols*elemSize, rows,
                             hipMemcpyHostToDevice,
                             stream );
    POP_CUDA_FATAL_TEST( err, "Failed to copy 2D plane host-to-device: " );
}

__host__
void PlaneBase::memcpyToHost( void* dst, int dst_pitch,
                              void* src, int src_pitch,
                              short cols, short rows,
                              int elemSize )
{
    assert( dst != 0 );
    assert( dst_pitch != 0 );
    assert( src != 0 );
    assert( src_pitch != 0 );
    assert( cols != 0 );
    assert( rows != 0 );
    hipError_t err;
    err = hipMemcpy2D( dst, dst_pitch,
                        src, src_pitch,
                        cols*elemSize, rows,
                        hipMemcpyDeviceToHost );
    POP_CUDA_FATAL_TEST( err, "Failed to copy 2D plane device-to-host: " );
}

__host__
void PlaneBase::memcpyToHost( void* dst, int dst_pitch,
                              void* src, int src_pitch,
                              short cols, short rows,
                              int elemSize,
                              hipStream_t stream )
{
    assert( dst != 0 );
    assert( dst_pitch != 0 );
    assert( src != 0 );
    assert( src_pitch != 0 );
    assert( cols != 0 );
    assert( rows != 0 );
    hipError_t err;
    err = hipMemcpy2DAsync( dst, dst_pitch,
                             src, src_pitch,
                             cols*elemSize, rows,
                             hipMemcpyDeviceToHost,
                             stream );
    POP_CUDA_FATAL_TEST( err, "Failed to copy 2D plane device-to-host: " );
}

#ifdef PLANE2D_CUDA_OP_DEBUG
__host__
void PlaneBase::waitAndCheck( hipStream_t stream ) const
{
    hipStreamSynchronize( stream );
    hipError_t err = hipGetLastError( );
    POP_CUDA_FATAL_TEST( err, "Failed in error check after async 2D plane operation: " );
}
#endif // PLANE2D_CUDA_OP_DEBUG

} // namespace popsift

